#include "hip/hip_runtime.h"
#include <PPFMap/Map.h>
#include <PPFMap/PPFEstimationKernel.h>

__constant__ float alignment_transformation[12];


ppfmap::Map::Map(const pcl::cuda::PointCloudSOA<pcl::cuda::Host>::Ptr cloud,
                 const pcl::cuda::PointCloudSOA<pcl::cuda::Host>::Ptr normals,
                 const float disc_dist,
                 const float disc_angle)
    : discretization_distance(disc_dist)
    , discretization_angle(disc_angle) {

    const size_t number_of_points = cloud->size();

    pcl::cuda::PointCloudSOA<pcl::cuda::Device> d_cloud;
    pcl::cuda::PointCloudSOA<pcl::cuda::Device> d_normals;

    d_cloud << *cloud;
    d_normals << *normals;

    ppf_codes.resize(number_of_points * number_of_points);

    for (int i = 0; i < number_of_points; i++) {
    
        const float3 point_position = make_float3(cloud->points_x[i],
                                                  cloud->points_y[i],
                                                  cloud->points_z[i]);

        const float3 point_normal = make_float3(normals->points_x[i],
                                                normals->points_y[i],
                                                normals->points_z[i]);

        // Calculate the angle between the normal and the X axis.
        float rotation_angle = acosf(point_normal.x);

        // Rotation axis lays on the plane y-z (i.e. u = 0)
        float v;
        float w;

        // The rotation axis is the cross product of the normal and the X axis. 
        if (point_normal.y == 0.0f && point_normal.z == 0.0f) {
            // Degenerate case, set the Y axis as the rotation axis
            v = 1.0f;
            w = 0.0f;
        } else {
            // This would be the cross product of the normal and the x axis.
            v = point_normal.z;
            w = - point_normal.y;
        }

        // Normalize vector
        float norm = sqrt(v * v + w * w);
        v /= norm;
        w /= norm;

        float affine[12];

        // First row of rotation matrix
        affine[0] = (v * v + w * w) * cosf(rotation_angle); 
        affine[1] = - w * sinf(rotation_angle); 
        affine[2] = v * sinf(rotation_angle); 

        // Second row of rotation matrix
        affine[4] = w * sinf(rotation_angle);
        affine[5] = v * v + w * w * cosf(rotation_angle); 
        affine[6] = v * w * (1.0f - cosf(rotation_angle)); 

        // Third row of rotation matrix
        affine[8] = - v * sinf(rotation_angle);
        affine[9] = v * w * (1.0f - cosf(rotation_angle)); 
        affine[10] = w * w + v * v * cosf(rotation_angle); 

        // Translation column
        affine[3] = - point_position.x * affine[0] 
                    - point_position.y * affine[1] 
                    - point_position.z * affine[2];

        affine[7] = - point_position.x * affine[4] 
                    - point_position.y * affine[5] 
                    - point_position.z * affine[6];

        affine[11] = - point_position.x * affine[8] 
                     - point_position.y * affine[9] 
                     - point_position.z * affine[10];

        // Set the transformation to the constant memory of the gpu.
        hipMemcpyToSymbol(HIP_SYMBOL(alignment_transformation), affine, 12 * sizeof(float));

        ppfmap::PPFEstimationKernel<pcl::cuda::Device> ppfe(point_position,
                                                            point_normal,
                                                            i,
                                                            discretization_distance,
                                                            discretization_angle,
                                                            alignment_transformation);

        thrust::transform(d_cloud.zip_begin(), d_cloud.zip_end(),
                          d_normals.zip_begin(),
                          ppf_codes.begin() + i * cloud->size(),
                          ppfe);

    }

    for (int i = 0; i < ppf_codes.size(); i++) {
        const uint64_t code = ppf_codes[i];

        uint32_t hk = static_cast<uint32_t>(code >> 32);
        uint32_t id = static_cast<uint32_t>(code >> 16 & 0xFFFF);
        uint32_t angle = static_cast<uint32_t>(code & 0xFFFF);

        std::cout << hk << " | " << id << " | " << angle << std::endl;
    }
}
