#include "hip/hip_runtime.h"
#include <PPFMap/Map.h>
#include <PPFMap/PPFEstimationKernel.h>


struct extract_hash_key : thrust::unary_function<uint64_t, uint32_t> {
    __host__ __device__
    uint32_t operator()(const uint64_t ppf_code) const {
        return static_cast<uint32_t>(ppf_code >> 32);
    }
};


ppfmap::Map::Map(const pcl::cuda::PointCloudSOA<pcl::cuda::Host>::Ptr cloud,
                 const pcl::cuda::PointCloudSOA<pcl::cuda::Host>::Ptr normals,
                 const float disc_dist,
                 const float disc_angle)
    : discretization_distance(disc_dist)
    , discretization_angle(disc_angle) {

    const size_t number_of_points = cloud->size();
    const size_t number_of_pairs = number_of_points * number_of_points;

    pcl::cuda::PointCloudSOA<pcl::cuda::Device> d_cloud;
    pcl::cuda::PointCloudSOA<pcl::cuda::Device> d_normals;

    d_cloud << *cloud;
    d_normals << *normals;

    ppf_codes.resize(number_of_pairs);

    for (int i = 0; i < number_of_points; i++) {
    
        const float3 point_position = make_float3(cloud->points_x[i],
                                                  cloud->points_y[i],
                                                  cloud->points_z[i]);

        const float3 point_normal = make_float3(normals->points_x[i],
                                                normals->points_y[i],
                                                normals->points_z[i]);

        // Calculate the angle between the normal and the X axis.
        float rotation_angle = acosf(point_normal.x);

        // Rotation axis lays on the plane y-z (i.e. u = 0)
        float v;
        float w;

        // The rotation axis is the cross product of the normal and the X axis. 
        if (point_normal.y == 0.0f && point_normal.z == 0.0f) {
            // Degenerate case, set the Y axis as the rotation axis
            v = 1.0f;
            w = 0.0f;
        } else {
            // This would be the cross product of the normal and the x axis.
            v = point_normal.z;
            w = - point_normal.y;
        }

        // Normalize vector
        float norm = sqrt(v * v + w * w);
        v /= norm;
        w /= norm;

        float affine[12];

        // First row of rotation matrix
        affine[0] = (v * v + w * w) * cosf(rotation_angle); 
        affine[1] = - w * sinf(rotation_angle); 
        affine[2] = v * sinf(rotation_angle); 

        // Second row of rotation matrix
        affine[4] = w * sinf(rotation_angle);
        affine[5] = v * v + w * w * cosf(rotation_angle); 
        affine[6] = v * w * (1.0f - cosf(rotation_angle)); 

        // Third row of rotation matrix
        affine[8] = - v * sinf(rotation_angle);
        affine[9] = v * w * (1.0f - cosf(rotation_angle)); 
        affine[10] = w * w + v * v * cosf(rotation_angle); 

        // Translation column
        affine[3] = - point_position.x * affine[0] 
                    - point_position.y * affine[1] 
                    - point_position.z * affine[2];

        affine[7] = - point_position.x * affine[4] 
                    - point_position.y * affine[5] 
                    - point_position.z * affine[6];

        affine[11] = - point_position.x * affine[8] 
                     - point_position.y * affine[9] 
                     - point_position.z * affine[10];

        ppfmap::PPFEstimationKernel<pcl::cuda::Device> 
            ppfe(point_position, point_normal, i,
                 discretization_distance,
                 discretization_angle,
                 affine);

        thrust::transform(d_cloud.zip_begin(), d_cloud.zip_end(),
                          d_normals.zip_begin(),
                          ppf_codes.begin() + i * cloud->size(),
                          ppfe);
    }

    thrust::sort(ppf_codes.begin(), ppf_codes.end());

    thrust::device_vector<uint32_t> hash_tmp(number_of_pairs);

    // copy the hash keys to a separate vector
    thrust::transform(
        ppf_codes.begin(), 
        ppf_codes.end(), 
        hash_tmp.begin(),
        extract_hash_key()
    );

    hash_keys.resize(number_of_pairs);
    ppf_count.resize(number_of_pairs);

    thrust::pair<thrust::device_vector<uint32_t>::iterator, 
                 thrust::device_vector<uint32_t>::iterator> end;

    // Count the number of similar keys
    end = thrust::reduce_by_key(hash_tmp.begin(), hash_tmp.end(),
                                thrust::make_constant_iterator(1),
                                hash_keys.begin(),
                                ppf_count.begin());

    const size_t unique_hash_keys = end.first - hash_keys.begin();

    // Fix the vectors to size
    hash_keys.resize(unique_hash_keys);
    hash_keys.shrink_to_fit();
    ppf_count.resize(unique_hash_keys);
    ppf_count.shrink_to_fit();

    ppf_index.resize(unique_hash_keys);

    // Set the array with the indices to the first instance of each key in the 
    // codes array.
    thrust::exclusive_scan(ppf_count.begin(), ppf_count.end(),
                           ppf_index.begin());

    max_votes = thrust::reduce(ppf_count.begin(), ppf_count.end(), 
                               0, thrust::maximum<uint32_t>());
}
