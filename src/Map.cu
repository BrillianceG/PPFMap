#include "hip/hip_runtime.h"
#include <PPFMap/Map.h>
#include <PPFMap/PPFEstimationKernel.h>


struct extract_hash_key : thrust::unary_function<uint64_t, uint32_t> {
    __host__ __device__
    uint32_t operator()(const uint64_t ppf_code) const {
        return static_cast<uint32_t>(ppf_code >> 32);
    }
};


struct copy_element_by_index : thrust::unary_function<uint32_t, uint32_t> {
    const uint32_t* ppf_index_ptr;

    copy_element_by_index(thrust::device_vector<uint32_t> const& vec) 
        : ppf_index_ptr(thrust::raw_pointer_cast(vec.data())) {}

    __host__ __device__
    uint32_t operator()(const uint32_t index) const {
        return ppf_index_ptr[index];
    }
};


struct write_votes {
    const uint64_t* model_ppf_ptr;
    const float discretization_angle;
    uint32_t* votes_ptr;

    write_votes(thrust::device_vector<uint64_t> const& model_ppf,
                const float disc_angle,
                thrust::device_vector<uint32_t> &votes)
        : model_ppf_ptr(thrust::raw_pointer_cast(model_ppf.data()))
        , discretization_angle(disc_angle)
        , votes_ptr(thrust::raw_pointer_cast(votes.data())) {}

    template <class Tuple> __device__
    void operator()(Tuple t) {

        const uint32_t insert_position = thrust::get<0>(t); 
        const bool     key_found = thrust::get<1>(t); 
        const uint32_t ppf_index = thrust::get<2>(t); 
        const uint32_t ppf_count = thrust::get<3>(t); 
        const float alpha_s = thrust::get<4>(t); 

        if (key_found) {
            for (int vote_idx = 0; vote_idx < ppf_count; vote_idx++) {

                uint64_t model_ppf_code = model_ppf_ptr[ppf_index + vote_idx];

                uint16_t model_index = static_cast<uint16_t>(model_ppf_code >> 16 & 0xFFFF);
                float alpha_m = static_cast<float>(model_ppf_code & 0xFFFF) * discretization_angle;

                uint16_t alpha = static_cast<uint16_t>((alpha_m - alpha_s) / discretization_angle);

                uint32_t vote = static_cast<uint32_t>(model_index) << 16 |
                                static_cast<uint32_t>(alpha);

                votes_ptr[insert_position + vote_idx] =  vote;
            }
        }
    }
};


ppfmap::Map::Map(const pcl::cuda::PointCloudSOA<pcl::cuda::Host>::Ptr cloud,
                 const pcl::cuda::PointCloudSOA<pcl::cuda::Host>::Ptr normals,
                 const float disc_dist,
                 const float disc_angle)
    : discretization_distance(disc_dist)
    , discretization_angle(disc_angle) {

    const size_t number_of_points = cloud->size();
    const size_t number_of_pairs = number_of_points * number_of_points;

    float affine[12];

    pcl::cuda::PointCloudSOA<pcl::cuda::Device> d_cloud;
    pcl::cuda::PointCloudSOA<pcl::cuda::Device> d_normals;

    d_cloud << *cloud;
    d_normals << *normals;

    ppf_codes.resize(number_of_pairs);

    for (int i = 0; i < number_of_points; i++) {
    
        const float3 point_position = make_float3(cloud->points_x[i],
                                                  cloud->points_y[i],
                                                  cloud->points_z[i]);

        const float3 point_normal = make_float3(normals->points_x[i],
                                                normals->points_y[i],
                                                normals->points_z[i]);

        ppfmap::getAlignmentToX(point_position, point_normal, &affine);

        ppfmap::PPFEstimationKernel<pcl::cuda::Device> 
            ppfe(point_position, point_normal, i,
                 discretization_distance,
                 discretization_angle,
                 affine);

        thrust::transform(d_cloud.zip_begin(), d_cloud.zip_end(),
                          d_normals.zip_begin(),
                          ppf_codes.begin() + i * cloud->size(),
                          ppfe);
    }

    thrust::sort(ppf_codes.begin(), ppf_codes.end());

    thrust::device_vector<uint32_t> hash_tmp(number_of_pairs);

    // copy the hash keys to a separate vector
    thrust::transform(
        ppf_codes.begin(), 
        ppf_codes.end(), 
        hash_tmp.begin(),
        extract_hash_key()
    );

    hash_keys.resize(number_of_pairs);
    ppf_count.resize(number_of_pairs);

    thrust::pair<thrust::device_vector<uint32_t>::iterator, 
                 thrust::device_vector<uint32_t>::iterator> end;

    // Count the number of similar keys
    end = thrust::reduce_by_key(hash_tmp.begin(), hash_tmp.end(),
                                thrust::make_constant_iterator(1),
                                hash_keys.begin(),
                                ppf_count.begin());

    const size_t unique_hash_keys = end.first - hash_keys.begin();

    // Fix the vectors to size
    hash_keys.resize(unique_hash_keys);
    hash_keys.shrink_to_fit();
    ppf_count.resize(unique_hash_keys);
    ppf_count.shrink_to_fit();

    ppf_index.resize(unique_hash_keys);

    // Set the array with the indices to the first instance of each key in the 
    // codes array.
    thrust::exclusive_scan(ppf_count.begin(), ppf_count.end(),
                           ppf_index.begin());

    max_votes = thrust::reduce(ppf_count.begin(), ppf_count.end(), 
                               0, thrust::maximum<uint32_t>());
}


void ppfmap::Map::searchBestMatch(const thrust::host_vector<uint32_t> hash_list, 
                                  const thrust::host_vector<float> alpha_s_list,
                                  int& m_idx, float& alpha) {


    thrust::device_vector<uint32_t> d_hash_list = hash_list;
    thrust::device_vector<float> d_alpha_s_list = alpha_s_list;

    thrust::device_vector<bool> d_key_found(d_hash_list.size());
    thrust::device_vector<uint32_t> d_key_index(d_hash_list.size());
    thrust::device_vector<uint32_t> d_ppf_index(d_hash_list.size());
    thrust::device_vector<uint32_t> d_ppf_count(d_hash_list.size());
    thrust::device_vector<uint32_t> d_insert_pos(d_hash_list.size());

    thrust::binary_search(hash_keys.begin(), hash_keys.end(),
                          d_hash_list.begin(), d_hash_list.end(),
                          d_key_found.begin());

    thrust::lower_bound(hash_keys.begin(), hash_keys.end(),
                        d_hash_list.begin(), d_hash_list.end(),
                        d_key_index.begin());

    thrust::transform(d_key_index.begin(), d_key_index.end(), 
                      d_ppf_index.begin(), 
                      copy_element_by_index(ppf_index));

    thrust::transform(d_key_index.begin(), d_key_index.end(), d_ppf_count.begin(), 
                      copy_element_by_index(ppf_count));

    uint64_t votes_total = thrust::reduce(d_ppf_count.begin(), d_ppf_count.end(), 
                                          0, thrust::plus<uint64_t>());

    // This sets the position where to start inserting the votes of each ppf
    thrust::exclusive_scan(d_ppf_count.begin(), d_ppf_count.end(), d_insert_pos.begin());

    thrust::device_vector<uint32_t> votes(votes_total);
    thrust::device_vector<uint32_t> unique_votes(votes_total);
    thrust::device_vector<uint32_t> vote_count(votes_total);

    thrust::for_each(
        thrust::make_zip_iterator(
            thrust::make_tuple(
                d_insert_pos.begin(), 
                d_key_found.begin(),
                d_ppf_index.begin(),
                d_ppf_count.begin(),
                d_alpha_s_list.begin()
            )
        ),          
        thrust::make_zip_iterator(
            thrust::make_tuple(
                d_insert_pos.end(), 
                d_key_found.begin(),
                d_ppf_index.end(),
                d_ppf_count.end(),
                d_alpha_s_list.end()
            )
        ),          
        write_votes(ppf_codes, discretization_angle, votes)
    );

    thrust::sort(votes.begin(), votes.end());

    thrust::pair<thrust::device_vector<uint32_t>::iterator, 
                 thrust::device_vector<uint32_t>::iterator> end;

    end = thrust::reduce_by_key(votes.begin(), votes.end(), 
                                thrust::make_constant_iterator(1), 
                                unique_votes.begin(), 
                                vote_count.begin());

    unique_votes.resize(end.first - unique_votes.begin());
    vote_count.resize(end.second - vote_count.begin());

    thrust::device_vector<uint32_t>::iterator iter =
          thrust::max_element(vote_count.begin(), vote_count.end());

    int position = iter - vote_count.begin();

    m_idx = static_cast<int>(unique_votes[position] >> 16);
    alpha = static_cast<float>(unique_votes[position] & 0xFFFF) * discretization_angle;
}
