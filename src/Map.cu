#include "hip/hip_runtime.h"
#include <PPFMap/Map.h>
#include <PPFMap/PPFEstimationKernel.h>


struct extract_hash_key : public thrust::unary_function<uint64_t, uint32_t> {
    __host__ __device__
    uint32_t operator()(const uint64_t ppf_code) const {
        return static_cast<uint32_t>(ppf_code >> 32);
    }
};


struct copy_element_by_index : public thrust::unary_function<uint32_t, uint32_t> {
    const uint32_t* ppf_index_ptr;

    copy_element_by_index(thrust::device_vector<uint32_t> const& vec) 
        : ppf_index_ptr(thrust::raw_pointer_cast(vec.data())) {}

    __host__ __device__
    uint32_t operator()(const uint32_t index) const {
        return ppf_index_ptr[index];
    }
};


struct VotesExtraction {
    const float discretization_angle;

    const float* alpha_s;

    const uint64_t* ppf_codes;
    const bool* ppf_found;
    const uint32_t* ppf_index;
    const uint32_t* ppf_count;
    const uint32_t* insert;

    uint32_t* votes_ptr;

    VotesExtraction(const thrust::device_vector<float>& alphas,
                    const thrust::device_vector<uint64_t>& map_codes,
                    const thrust::device_vector<bool>& map_found,
                    const thrust::device_vector<uint32_t>& map_index,
                    const thrust::device_vector<uint32_t>& map_count,
                    const thrust::device_vector<uint32_t>& insert_votes,
                    const float disc_angle,
                    thrust::device_vector<uint32_t>& votes)
        : alpha_s(thrust::raw_pointer_cast(alphas.data()))
        , ppf_codes(thrust::raw_pointer_cast(map_codes.data()))
        , ppf_found(thrust::raw_pointer_cast(map_found.data()))
        , ppf_index(thrust::raw_pointer_cast(map_index.data()))
        , ppf_count(thrust::raw_pointer_cast(map_count.data()))
        , insert(thrust::raw_pointer_cast(insert_votes.data()))
        , discretization_angle(disc_angle)
        , votes_ptr(thrust::raw_pointer_cast(votes.data())) {}

    __device__
    void operator()(const int i) {
        if (ppf_found[i]) {
            for (int vote_idx = 0; vote_idx < ppf_count[i]; vote_idx++) {

                uint64_t model_ppf_code = ppf_codes[ppf_index[i] + vote_idx];

                uint16_t model_index = static_cast<uint16_t>(model_ppf_code >> 16 & 0xFFFF);
                float alpha_m = static_cast<float>(model_ppf_code & 0xFFFF) * discretization_angle;

                uint16_t alpha = static_cast<uint16_t>((alpha_m - alpha_s[i]) / discretization_angle);

                uint32_t vote = static_cast<uint32_t>(model_index) << 16 |
                                static_cast<uint32_t>(alpha);

                votes_ptr[insert[i] + vote_idx] =  vote;
            }
        }
    }
};


struct PPFMapSearch {
    const std::size_t n;
    const uint32_t* hash_list;

    const uint32_t* hash_keys;
    const uint32_t* ppf_index;
    const uint32_t* ppf_count;

    bool* out_found;
    uint32_t* out_index;
    uint32_t* out_count;

    PPFMapSearch(const thrust::device_vector<uint32_t>& hl,
                 const thrust::device_vector<uint32_t>& map_hash_keys,
                 const thrust::device_vector<uint32_t>& map_ppf_index,
                 const thrust::device_vector<uint32_t>& map_ppf_count,
                 thrust::device_vector<bool>& result_found,
                 thrust::device_vector<uint32_t>& result_index,
                 thrust::device_vector<uint32_t>& result_count)
        : n(map_hash_keys.size())
        , hash_list(thrust::raw_pointer_cast(hl.data()))
        , hash_keys(thrust::raw_pointer_cast(map_hash_keys.data()))
        , ppf_index(thrust::raw_pointer_cast(map_ppf_index.data()))
        , ppf_count(thrust::raw_pointer_cast(map_ppf_count.data()))
        , out_found(thrust::raw_pointer_cast(result_found.data()))
        , out_index(thrust::raw_pointer_cast(result_index.data()))
        , out_count(thrust::raw_pointer_cast(result_count.data())) {}

    __device__
    void operator()(const int i) {
        const uint32_t hk = hash_list[i]; 

        out_found[i] = false;
        out_index[i] = 0;
        out_count[i] = 0;
    
        int l = 0;
        int r = n;
        int m = (l + r) / 2;

        while (l < r) {
            if (hk < hash_keys[m]) {
                r = m;
            }
            else if (hk > hash_keys[m]) {
                l = m + 1;
            }
            else {
                out_found[i] = true;
                out_index[i] = ppf_index[m];
                out_count[i] = ppf_count[m];
                break; 
            } 
            m = (l + r) / 2; 
        }
    }
};


/** \brief Computes the PPF features for the input cloud.
 *  \param[in] h_points Host vector with the 3D information of the points.
 *  \param[in] h_normals Host vector with the normals of each point.
 *  \param[in] disc_dist Discretization factor for pair distance.
 *  \param[in] disc_angle Discretization factor for angles.
 */
ppfmap::Map::Map(const pcl::cuda::Host<float3>::type& h_points,
                 const pcl::cuda::Host<float3>::type& h_normals,
                 const float disc_dist,
                 const float disc_angle)
    : discretization_distance(disc_dist)
    , discretization_angle(disc_angle) {

    const std::size_t number_of_points = h_points.size();
    const std::size_t number_of_pairs = number_of_points * number_of_points;

    float affine[12];

    pcl::cuda::Device<float3>::type d_points(h_points);
    pcl::cuda::Device<float3>::type d_normals(h_normals);

    ppf_codes.resize(number_of_pairs);

    float max_distance = 0.0f;
    for (int i = 0; i < number_of_points; i++) {
        const float3 point_position = h_points[i];
        const float3 point_normal = h_normals[i];

        ppfmap::getAlignmentToX(point_position, point_normal, &affine);

        ppfmap::PPFEstimationKernel ppfe(point_position, point_normal, i,
                                         discretization_distance,
                                         discretization_angle,
                                         affine);

        thrust::transform(d_points.begin(), d_points.end(),
                          d_normals.begin(),
                          ppf_codes.begin() + i * number_of_points,
                          ppfe);

        float max_pair_dist = ppfmap::maxDistanceToPoint<pcl::cuda::Device>(point_position, d_points);

        if (max_distance < max_pair_dist) {
            max_distance = max_pair_dist; 
        }
    }
    
    cloud_diameter = max_distance;

    thrust::sort(ppf_codes.begin(), ppf_codes.end());

    thrust::device_vector<uint32_t> hash_tmp(number_of_pairs);

    // copy the hash keys to a separate vector
    thrust::transform(
        ppf_codes.begin(), 
        ppf_codes.end(), 
        hash_tmp.begin(),
        extract_hash_key()
    );

    hash_keys.resize(number_of_pairs);
    ppf_count.resize(number_of_pairs);

    thrust::pair<thrust::device_vector<uint32_t>::iterator, 
                 thrust::device_vector<uint32_t>::iterator> end;

    // Count the number of similar keys
    end = thrust::reduce_by_key(hash_tmp.begin(), hash_tmp.end(),
                                thrust::make_constant_iterator(1),
                                hash_keys.begin(),
                                ppf_count.begin());

    const size_t unique_hash_keys = end.first - hash_keys.begin();

    // Fix the vectors to size
    hash_keys.resize(unique_hash_keys);
    hash_keys.shrink_to_fit();
    ppf_count.resize(unique_hash_keys);
    ppf_count.shrink_to_fit();

    ppf_index.resize(unique_hash_keys);

    // Set the array with the indices to the first instance of each key in the 
    // codes array.
    thrust::exclusive_scan(ppf_count.begin(), ppf_count.end(),
                           ppf_index.begin());

    max_votes = thrust::reduce(ppf_count.begin(), ppf_count.end(), 
                               0, thrust::maximum<uint32_t>());
}


/** \brief Performs the voting and accumulation for the ppf list provided 
 * and returns the best point index and resulting alpha.
 *  \param[in] hash_list List of hashed ppf features to query
 *  \param[in] alpha_s Angle to align the reference point to the x axis.
 *  \param[out] m_idx Best matching index in Hough voting space.
 *  \param[out] alpha Resulting angle after combining the alpha_s and 
 *  alpha_m.
 *  \param[out] max_votes The number of pairs supporting the m_idx and alpha 
 *  parameters.
 */
void ppfmap::Map::searchBestMatch(const thrust::host_vector<uint32_t> hash_list, 
                                  const thrust::host_vector<float> alpha_s_list,
                                  int& m_idx, float& alpha, int& max_votes) {


    thrust::device_vector<uint32_t> d_hash_list(hash_list);
    thrust::device_vector<float> d_alpha_s_list(alpha_s_list);

    thrust::device_vector<bool> d_key_found(d_hash_list.size());
    thrust::device_vector<uint32_t> d_ppf_index(d_hash_list.size());
    thrust::device_vector<uint32_t> d_ppf_count(d_hash_list.size());
    thrust::device_vector<uint32_t> d_insert_pos(d_hash_list.size());

    PPFMapSearch m_search(d_hash_list, 
                          hash_keys, ppf_index, ppf_count, 
                          d_key_found, d_ppf_index, d_ppf_count);

    thrust::counting_iterator<int> it(0);
    thrust::for_each(it, it + hash_list.size(), m_search);

    uint64_t votes_total = thrust::reduce(d_ppf_count.begin(), d_ppf_count.end(), 
                                          0, thrust::plus<uint32_t>());

    // This sets the position where to start inserting the votes of each ppf
    thrust::exclusive_scan(d_ppf_count.begin(), d_ppf_count.end(), d_insert_pos.begin());

    thrust::device_vector<uint32_t> votes(votes_total);
    thrust::device_vector<uint32_t> unique_votes(votes_total);
    thrust::device_vector<uint32_t> vote_count(votes_total);

    VotesExtraction write_votes(d_alpha_s_list, 
                                ppf_codes, 
                                d_key_found, d_ppf_index, 
                                d_ppf_count, d_insert_pos,
                                discretization_angle,
                                votes);

    thrust::for_each(it, it + hash_list.size(), write_votes);

    thrust::sort(votes.begin(), votes.end());

    thrust::pair<thrust::device_vector<uint32_t>::iterator, 
                 thrust::device_vector<uint32_t>::iterator> end;

    end = thrust::reduce_by_key(votes.begin(), votes.end(), 
                                thrust::make_constant_iterator(1), 
                                unique_votes.begin(), 
                                vote_count.begin());

    unique_votes.resize(end.first - unique_votes.begin());
    vote_count.resize(end.second - vote_count.begin());

    thrust::device_vector<uint32_t>::iterator iter =
          thrust::max_element(vote_count.begin(), vote_count.end());

    int position = iter - vote_count.begin();

    const uint32_t winner = unique_votes[position];

    m_idx = static_cast<int>(winner >> 16);
    alpha = static_cast<float>(winner & 0xFFFF) * discretization_angle;
    max_votes = static_cast<int>(*iter);
}
