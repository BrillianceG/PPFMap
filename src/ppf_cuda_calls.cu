#include "hip/hip_runtime.h"
#include <PPFMap/ppf_cuda_calls.h>

ppfmap::Map::Ptr 
ppfmap::cuda::setPPFMap(const float3 *points, 
                        const float3 *normals,
                        const size_t n,
                        const float disc_dist,
                        const float disc_angle) {

    pcl::cuda::Host<float3>::type h_points(points, points + n);
    pcl::cuda::Host<float3>::type h_normals(normals, normals + n);

    return boost::shared_ptr<Map>(new Map(h_points, h_normals, disc_dist, disc_angle));
}
